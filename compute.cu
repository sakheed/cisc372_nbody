#include "hip/hip_runtime.h"
//Sakhee Desai and Aman Singh
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "vector.h"
#include "config.h"

__global__
void computePairwiseAccels(v3 *acceleration, v3 *position, double *mass){
	int k;
	v3 dist;

	int i = blockIdx.x*blockDim.x + threadIdx.x;
   	int j = blockIdx.y*blockDim.y + threadIdx.y;
	
	double magnitude, magA, mag_square;

	if(i < NUMENTITIES && j < NUMENTITIES){
			if (i==j) {
				FILL_VECTOR(acceleration[i*NUMENTITIES+j],0,0,0);
			}
			else{
				for (k=0;k<3;k++) {
					dist[k]= position[i][k]- position[j][k];
				}
				mag_square = dist[0]*dist[0]+dist[1]*dist[1]+dist[2]*dist[2];
				magnitude = sqrt(mag_square);
				magA = -1 * GRAV_CONSTANT * mass[j]/mag_square;
				FILL_VECTOR(acceleration[i*NUMENTITIES +j],
						magA*dist[0]/magnitude,	
						magA*dist[1]/magnitude,
						magA*dist[2]/magnitude);
			}
	}
}

__global__
void sumRowsandUpdate(v3* acceleration, v3* velocity, v3* position, double* mass){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
        int j,k;
	v3 total_acceleration = {0,0,0};	
	if (i<NUMENTITIES){
		for (j=0;j<NUMENTITIES;j++){
			for (k=0;k<3;k++){
				total_acceleration[k]+=acceleration[i*NUMENTITIES + j][k];
			}
		}
		 for (k=0;k<3;k++){
                  	velocity[i][k] += total_acceleration[k]*INTERVAL;
                  	position[i][k] += velocity[i][k]*INTERVAL;
          	}
	}
}

//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL

void compute(){
	new_dim szBlk(16,16);
	new_dim nBlk((NUMENTITIES*NUMENTITIES +szBlk.x-1)/szBlk.x, (NUMENTITIES*NUMENTITIES +szBlk.y-1)/szBlk.y);
	int new_BS = 256;	
	int new_num_blocks = (NUMENTITIES*NUMENTITIES + newBS - 1)/newBS;

	computePairwiseAccels<<<nBlk, szBlk>>>(acceleration, position, new_mass);
	sumRowsandUpdate<<<new_num_blocks, newBS>>>(acceleration, velocity, position, new_mass);	
	
}
