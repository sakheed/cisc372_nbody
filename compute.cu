#include "hip/hip_runtime.h"
//Sakhee Desai and Aman Singh
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "vector.h"
#include "config.h"

__global__ void acceleration_computation(v3 *acceleration, v3 *position, double *mass){
	int k;
	int l;
	v3 dist;

	int i = blockIdx.x*blockDim.x + threadIdx.x;
   	int j = blockIdx.y*blockDim.y + threadIdx.y;
	
	double magnitude;
	double magA;
	double mag_square;

	if(i < NUMENTITIES && j < NUMENTITIES){
			if (i==j) {
				FILL_VECTOR(acceleration[NUMENTITIES * i + j], 0, 0, 0);
			} else {
				for (k = 0; k < 3; k++) dist[k] = position[i][k] - position[j][k];
				for (l = 0; l < 3; l++) mag_square += dist[l] * dist[l];

				magnitude = sqrt(mag_square);
				magA = GRAV_CONSTANT * (mass[j] / mag_square) * -1;

				FILL_VECTOR(acceleration[NUMENTITIES * i + j],
						dist[0] * magA / magnitude,	
						dist[1] * magA / magnitude,
						dist[2] * magA / magnitude);
			}
	}
}

__global__ void row_summation(v3* acceleration, v3* velocity, v3* position, double* mass){
	int i = threadIdx.x + (blockIdx.x * blockDim.x);
        int j;
	int k;

	v3 total_acceleration = {0, 0, 0};

	if (i < NUMENTITIES) {
		for (j = 0; j < NUMENTITIES; j++) {
			for (k = 0; k < 3; k++) total_acceleration[k] += acceleration[j + NUMENTITIES * i][k];
		}
		for (k = 0; k < 3; k++) {
                  	velocity[i][k] += INTERVAL * total_acceleration[k];
                  	position[i][k] += INTERVAL * velocity[i][k];
          	}
	}
}

//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL

void compute(){
	new_dim szBlk(16, 16);
	int new_BS = 256;
	new_dim nBlk((NUMENTITIES*NUMENTITIES + szBlk.x - 1) / szBlk.x, (NUMENTITIES * NUMENTITIES +szBlk.y - 1) / szBlk.y);
	int new_num_blocks = (NUMENTITIES*NUMENTITIES + newBS - 1) / newBS;

	acceleration_computation<<<nBlk, szBlk>>>(acceleration, position, new_mass);
	row_summation<<<new_num_blocks, newBS>>>(acceleration, velocity, position, new_mass);	
	
}
